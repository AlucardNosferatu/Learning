#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void hello_from_gpu()
{
    printf("Hello World!\n");
}

int main(void)
{
    hello_from_gpu<<<2, 2>>>();
    hipDeviceSynchronize();
    return 0;
}